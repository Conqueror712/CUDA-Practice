#include <stdio.h>
#include <dlfcn.h>
#include <hip/hip_runtime_api.h>

int main(){
    void *handle = dlopen("./libmycuda.so", RTLD_NOW);
    if (handle == NULL){
        fprintf(stderr, "Error: cannot load library: %s\n", dlerror());
        return 1;
    }
    hipError_t (*my_cudaMallocManaged)(void **, size_t, unsigned int) = dlsym(handle, "my_cudaMallocManaged");
    if (my_cudaMallocManaged == NULL){
        fprintf(stderr, "Error: cannot find function: %s\n", dlerror());
        dlclose(handle);
        return 1;
    }
    void *devPtr;
    size_t size = 1024;
    unsigned int flags = hipMemAttachGlobal;
    hipError_t result = my_cudaMallocManaged(&devPtr, size, flags);
    if (result != hipSuccess){
        fprintf(stderr, "Error: my_cudaMallocManaged failed: %s\n", hipGetErrorString(result));
        dlclose(handle);
        return 1;
    }
    printf("Allocated device memory at %p\n", devPtr);
    result = hipFree(devPtr);
    if (result != hipSuccess){
        fprintf(stderr, "Error: hipFree failed: %s\n", hipGetErrorString(result));
        dlclose(handle);
        return 1;
    }
    dlclose(handle);
    return 0;
}
